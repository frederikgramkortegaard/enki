#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../../../interpreter/eval.hpp"

// CUDA kernel - pure GPU computation
__global__ void greyscale_kernel(unsigned char* input, unsigned char* output, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_idx = idx * 3;
    
    if (idx < width * height) {
        unsigned char r = input[pixel_idx];
        unsigned char g = input[pixel_idx + 1];
        unsigned char b = input[pixel_idx + 2];
        
        unsigned char gray = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
        
        output[pixel_idx] = gray;
        output[pixel_idx + 1] = gray;
        output[pixel_idx + 2] = gray;
    }
}

// C++ wrapper that handles CUDA setup and calls the kernel
extern "C" Value greyscale_cuda(const std::vector<Value>& args) {
    if (args.size() != 1) {
        throw std::runtime_error("greyscale_cuda(image): expects image");
    }
    
    auto img_val = std::dynamic_pointer_cast<ImageValue>(args[0]);
    if (!img_val) {
        throw std::runtime_error("greyscale_cuda(image): expects image");
    }
    
    int width = img_val->width;
    int height = img_val->height;
    int data_size = width * height * 3;
    
    // CUDA memory management and kernel launch
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, data_size);
    hipMalloc(&d_output, data_size);
    
    hipMemcpy(d_input, img_val->data.data(), data_size, hipMemcpyHostToDevice);
    
    int block_size = 256;
    int grid_size = (width * height + block_size - 1) / block_size;
    greyscale_kernel<<<grid_size, block_size>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
    
    std::vector<uint8_t> output_data(data_size);
    hipMemcpy(output_data.data(), d_output, data_size, hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_output);
    
    return std::make_shared<ImageValue>(width, height, std::move(output_data), img_val->format);
}